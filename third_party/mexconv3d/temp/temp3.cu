#include <hipblas.h>
#include "mex.h"
#include "gpu/mxGPUArray.h"

#include "../src/wrapperMx.h"
#include "../src/wrapperBlas.h"
#include "../src/Timer.h"


matw makeit (xpuMxArrayTW& rhs) {
  matw A;
  A.beg = (float*) rhs.getDataBeg();
  A.H = rhs.getSizeAtDim(0);
  A.W = rhs.getSizeAtDim(1);

  return A;
}

matw create_Y_ (mwSize M, mwSize N) {
  mwSize sz[2];
  sz[0] = M; sz[1] = N;
  mxGPUArray* tmp = mxGPUCreateGPUArray(2, sz, mxSINGLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);

  matw Y;
  Y.beg = (float*)mxGPUGetData(tmp);
  Y.H = M;
  Y.W = N;

  mxGPUDestroyGPUArray(tmp);
  return Y;
}


void mexFunction(int no, mxArray       *vo[],
                 int ni, mxArray const *vi[])
{
  // input
  xpuMxArrayTW a;
  a.setMxArray( (mxArray*) vi[0]);
  matw convmat = makeit(a);

  xpuMxArrayTW b;
  b.setMxArray( (mxArray*) vi[1]);
  matw F_ = makeit(b);

  // output
  matw Y_ = create_Y_(convmat.H, F_.W);

  // do it
  mexPrintf("[%d %d] [%d %d]\n", convmat.H, convmat.W, F_.H, F_.W);

  const int N = 1500;
  mexPrintf("num inst = %d\n", N);

  hipblasHandle_t hd;
  hipblasCreate(&hd);

  Timer tm;
  double time = 0.0;

  for (int i = 0; i < N; ++i) {
    float alpha = 1.0;
    float beta = 0.0;

    tm.start();

    hipblasStatus_t st = hipblasSgemm(
      hd,
      HIPBLAS_OP_N, HIPBLAS_OP_N,
      (int)convmat.H, (int)F_.W, (int)convmat.W,
      &alpha,
      (float*)convmat.beg, (int)convmat.H,
      (float*)F_.beg, (int)F_.H,
      &beta,
      (float*)Y_.beg, (int)Y_.H);

    tm.stop();
    time += tm.getElapsedTimeInMilliSec();
    
  }

  mexPrintf("time = %f\n", time);
  
  return;
}