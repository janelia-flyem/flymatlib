#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "_maxpool3d_gpu.h"
#include "_cu_helper.cuh"


namespace {

//// thin wrappers 
struct tw_array5d { 
  float *beg;
  int sz[5];
  int HW, HWD;
  int nelem;
};

template<typename T>
struct tw_vec {
  T* beg;
  int sz;
};

struct fprop_impl {
  tw_array5d  X, Y;
  tw_vec<int> ind;
 
  int pool[3];
  int stride[3];
  int pad[6];
};

struct bprop_impl
{
  tw_vec<float>  dX, dY;
  tw_vec<int> ind;
};

//// kernel Impl
__device__ const float VERY_NEGATIVE_NUM = -1e20;

__device__ void ind2sub (int iElem, int sz[5], 
                         int &h, int &w, int &d, int &iVol) 
{
  int H   = sz[0];
  int HW  = H * sz[1];
  int HWD = HW * sz[2];

  iVol = iElem / HWD;
  iElem = iElem % HWD;

  d = iElem / HW;
  iElem = iElem % HW;

  w = iElem / H;

  h = iElem % H;
}

__global__ void kernel_fprop (fprop_impl impl) {
  int iElem = blockIdx.x * blockDim.x + threadIdx.x;

  if (iElem >= impl.Y.nelem) return;

  // subscript on Y
  int iY, jY, kY, iVol;
  ind2sub(iElem, impl.Y.sz,  iY,jY,kY,iVol);

  // init value for current Y
  float vmax = VERY_NEGATIVE_NUM;
  int   imax = -43.0;

  // set the window on X for current Y element (iElem); note the offset can be negative
  int xH   = impl.X.sz[0];
  int xHW  = impl.X.HW;
  int xHWD = impl.X.HWD;
  int xwin_offset[3];
  xwin_offset[0] = -(impl.pad[0]) + ( iY * impl.stride[0] ); 
  xwin_offset[1] = -(impl.pad[2]) + ( jY * impl.stride[1] );
  xwin_offset[2] = -(impl.pad[4]) + ( kY * impl.stride[2] );
  const float* const xwin_beg = impl.X.beg + 
                                xwin_offset[0] + 
                                xwin_offset[1]*xH + 
                                xwin_offset[2]*xHW +
                                iVol*xHWD;

  // inspect the window at X, get the max value
  for (int t = 0; t < impl.pool[2]; ++t) {     // X window dim3: depth
    int xt = t + xwin_offset[2];
    bool xtInRange = (xt>=0) && (xt<impl.X.sz[2]);

    for (int s = 0; s < impl.pool[1]; ++s) {   // X window dim2: width
      int xs = s + xwin_offset[1];
      bool xsInRange = (xs>=0) && (xs<impl.X.sz[1]);

      for (int r = 0; r < impl.pool[0]; ++r) { // X window dim1: height
        int xr = r + xwin_offset[0];
        bool xrInRange = (xr>=0) && (xr<impl.X.sz[0]);

        // if out of range: never collect the element
        if ( !(xtInRange && xsInRange && xrInRange) )
          continue;

        // collect the element: current x value
        float vx = *(xwin_beg + r + s*xH + t*xHW);
        if (vx >= vmax) { // found new max value?
          vmax = vx;
          imax = double( xr + xs*xH + xt*xHW + iVol*xHWD );
        } // if

      } // r
    } // s
  } // t

  // write to the target
  impl.Y.beg[iElem]   = vmax;
  impl.ind.beg[iElem] = imax + 1; // to Matlab 1-base
}

__global__ void kernel_bprop (bprop_impl impl) {
  int iY = blockIdx.x * blockDim.x + threadIdx.x;

  if (iY >= impl.dY.sz) return;

  int ix = int( impl.ind.beg[iY] );
  ix -= 1;

  // atomic Increment: there can be overlapping ix!
  atomicAdd( (impl.dX.beg + ix), impl.dY.beg[iY] );
}

} // namespace

//// impl of public methods
maxpool3d_gpu::maxpool3d_gpu()
{

}

maxpool3d_gpu::maxpool3d_gpu(const maxpool3d &obj)
{
  for (int i = 0; i < 6; ++i) pad[i]  = obj.pad[i];
  for (int i = 0; i < 3; ++i) pool[i] = obj.pool[i];
  for (int i = 0; i < 3; ++i) stride[i] = obj.stride[i];

  ind = obj.ind;
  X  = obj.X;
  dX = obj.dX;
  Y  = obj.Y;
  dY = obj.dY;

  ct = obj.ct;

}

void maxpool3d_gpu::fprop()
{
  // create output
  create_Y();
  create_ind();


  // set the impl struct and run it
  fprop_impl impl;
  // options
  for (int i = 0; i < 3; ++i) impl.pool[i] = pool[i];
  for (int i = 0; i < 6; ++i) impl.pad[i] = pad[i];
  for (int i = 0; i < 3; ++i) impl.stride[i] = stride[i];
  // input: X, device pointer
  impl.X.beg = (float*) X.getDataBeg();
  for (int i = 0; i < 5; ++i) impl.X.sz[i] = X.getSizeAtDim(i);
  impl.X.HW    = impl.X.sz[0] * impl.X.sz[1];
  impl.X.HWD   = impl.X.HW * impl.X.sz[2];
  impl.X.nelem = static_cast<int>( numel(X) );
  // output: Y, device pointer
  impl.Y.beg = (float*) Y.getDataBeg();
  for (int i = 0; i < 5; ++i) impl.Y.sz[i] = Y.getSizeAtDim(i);
  impl.Y.HW    = impl.Y.sz[0] * impl.Y.sz[1];
  impl.Y.HWD   = impl.Y.HW * impl.Y.sz[2];
  impl.Y.nelem = static_cast<int>( numel(Y) );
  // output: ind, device pointer
  impl.ind.beg = (int*) ind.getDataBeg();
  impl.ind.sz  = static_cast<int>( numel(ind) );


  // run
  int nelem = static_cast<int>( numel(Y) );
  kernel_fprop<<<ceil_divide(nelem, CU_NUM_THREADS), CU_NUM_THREADS>>>( impl );
}

void maxpool3d_gpu::bprop()
{
  // create dX at input port
  check_dY_ind();
  create_dX();


  // set the impl struct
  bprop_impl impl;
  //
  impl.dX.beg = (float*) dX.getDataBeg();
  impl.dX.sz  = static_cast<int>( numel(dX) );
  //
  impl.dY.beg = (float*) dY.getDataBeg();
  impl.dY.sz  = static_cast<int>( numel(dY) );
  //
  impl.ind.beg = (int*) ind.getDataBeg();
  impl.ind.sz  = static_cast<int>( numel(ind) );


  // run
  kernel_bprop <<<ceil_divide(impl.dY.sz, CU_NUM_THREADS), CU_NUM_THREADS>>>( impl );
}

