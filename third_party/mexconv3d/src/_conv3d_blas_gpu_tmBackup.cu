#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "_conv3d_blas_gpu.h"
#include "_cu_helper.h"
#include "logmsg.h"
#ifdef TM
#include "Timer.h"
#include <hipblas.h>
#endif // TM

namespace {
//// helpers for threads


//// helper: setting initial value
template<typename T>
__global__ void kernelSetZero (T* beg, mwSize len) {
  mwSize ind = blockIdx.x * blockDim.x + threadIdx.x;
  if (ind < len) beg[ind] = static_cast<T>(0);
}

template<typename T>
__global__ void kernelSetOne (T* beg, mwSize len) {
  mwSize ind = blockIdx.x * blockDim.x + threadIdx.x;
  if (ind < len) beg[ind] = static_cast<T>(1);
}

//// Impl of copying data back and forth for Vol and Convmat
typedef conv3d_blas_gpu::CpyVolConvmatImpl CpyImpl;

__device__ mwSize get_convmat_h (CpyImpl &ip, mwSize indCM) {
  return (indCM % ip.convmat.H);
}

__device__ mwSize get_convmat_w (CpyImpl &ip, mwSize indCM) {
  return (indCM / ip.convmat.H);
}

__device__ void get_subY (CpyImpl &ip, mwSize ind,  mwSize subY[3]) {
  mwSize HW = ip.szY[0] * ip.szY[1];
  mwSize H  = ip.szY[0];

  subY[2] = ind / HW;
  ind %= HW;

  subY[1] = ind / H;
  ind %= H;

  subY[0] = ind;
}

__device__ void get_win_offset4 (CpyImpl &ip, mwSize h_covnmat,  int64_T win_offset[3]) {
  mwSize subY[3];
  get_subY(ip, h_covnmat, subY);

  for (int i = 0; i < 3; ++i) 
    win_offset[i] = -static_cast<int64_T>(ip.pad[2*i]) + static_cast<int64_T>(subY[i] * ip.stride[i]);
  win_offset[3] = 0;
}

__device__ void get_win_sub4 (CpyImpl &ip, mwSize w_convmat,  mwSize win_sub[4]) {
  mwSize H   = ip.szF[0]; 
  mwSize HW  = H * ip.szF[1];
  mwSize HWD = HW * ip.szF[2];

  win_sub[3] = w_convmat / HWD;
  w_convmat %= HWD;

  win_sub[2] = w_convmat / HW;
  w_convmat %= HW;

  win_sub[1] = w_convmat / H;
  w_convmat %= H;

  win_sub[0] = w_convmat;
}

// return -1 if out of range (either underflow or overflow)
__device__ int64_T get_indVol (CpyImpl &ip, int64_T win_offset[4], mwSize win_sub[4]) {

  // the global subscript and guaranteed valid range
  int64_T vol_sub[4];
  for (int i = 0; i < 4; ++i) {
    vol_sub[i] = win_offset[i] + static_cast<int64_T>(win_sub[i]);
    if ( vol_sub[i] < 0 ) return -1; // underflow
    if ( vol_sub[i] >= ip.vol_i.sz[i] ) return -1; // overflow
  }

  mwSize H   = ip.vol_i.sz[0];
  mwSize HW  = H * ip.vol_i.sz[1];
  mwSize HWD = HW * ip.vol_i.sz[2];

  return static_cast<int64_T>(HWD*vol_sub[3] + HW*vol_sub[2] + H*vol_sub[1] + vol_sub[0]);
}

const int DIR_VOL_TO_CONVMAT   = 0; // nvcc does not support enum instantiation?
const int DIR_VOL_FROM_CONVMAT = 1;

template<int dir>
void __global__ kernelCpyVolConvmat (CpyImpl ip) {
  mwSize indCM = blockDim.x * blockIdx.x + threadIdx.x;
  if ( indCM >= (ip.convmat.H*ip.convmat.W) ) return;

  // fill h, w
  mwSize h = get_convmat_h(ip, indCM); // convmat dim1
  mwSize w = get_convmat_w(ip, indCM); // convmat dim2

  // h (convmat dim1) -> window's offset (starting point) on volume (win_offset[3] = 0 as volume dim4 all in!)
  int64_T win_offset[4]; // fill win_offset
  get_win_offset4(ip, h, win_offset);

  // w (convmat dim2) -> win_sub ( r,s,t,u the subscript within the window )
  mwSize win_sub[4]; // (r, s, t, u) 
  get_win_sub4(ip, w, win_sub);

  // win_offset[4] and win_sub[4] -> linear index, ind, on volume
  int64_T indVol = get_indVol(ip, win_offset, win_sub);
  
  // copy the data at indCM, indVol
  if (indVol < 0) {
    if (dir == DIR_VOL_TO_CONVMAT) 
      ip.convmat.beg[indCM] = 0.0; // pad zeros!
    //else: DIR_VOL_FROM_CONVMAT, do nothing
    return;
  }
  
  if (dir == DIR_VOL_TO_CONVMAT) // vol -> convmat
    ip.convmat.beg[indCM] = ip.vol_i.beg[indVol];
  else { // DIR_VOL_FROM_CONVMAT, vol <- convmat
    // ATOMIC increment: ip.vol_i.beg[indVol] += ip.convmat.beg[indCM]
    atomicAdd( (ip.vol_i.beg + indVol), ip.convmat.beg[indCM]);
  }
}

} // namespace


//// impl of public methods
conv3d_blas_gpu::conv3d_blas_gpu()
{

}

conv3d_blas_gpu::conv3d_blas_gpu(const conv3d& obj)
{
  for (int i = 0; i < 6; ++i) pad[i]  = obj.pad[i];
  for (int i = 0; i < 3; ++i) stride[i] = obj.stride[i];

  X  = obj.X;
  dX = obj.dX;
  Y  = obj.Y;
  dY = obj.dY;
  F  = obj.F;
  dF = obj.dF;
  B  = obj.B;
  dB = obj.dB;

  ct = obj.ct;
}

void conv3d_blas_gpu::fprop()
{
  create_Y();
  init_convmat();

#ifdef TM
  Timer tm;
#endif // TM

  try {
    // iterate over each training instance
    CpyVolConvmatImpl ip = make_initial_CpyVolConvmatImpl( X );
    mwSize N = X.getSizeAtDim(4);

#ifdef TM
    mexPrintf("num inst = %d \n", N);
    tm.start();
#endif

    for (mwSize i = 0; i < N; i++) {

      // make phiX: the convolution matrix
      vol_to_convmat(ip, X, i);
      

    } // for i
  } // try
  catch (const blas_ex& e) {
    free_convmat();
    throw conv3d_ex(e.what());
  }

#ifdef TM
  hipDeviceSynchronize();

  tm.stop();
  double te = tm.getElapsedTimeInMilliSec();

  mexPrintf("conv3d_blas_gpu::vol_to_convmat: %f\n", te);
#endif // TM

  free_convmat();
  //free_u();

#ifdef TM
  mexPrintf("\n");
#endif // TM
}

//void conv3d_blas_gpu::fprop()
//{
//  create_Y();
//  init_convmat();
//  init_u(); 
//
//#ifdef TM
//  Timer tm;
//  tm.start();
//
//  Timer tmtm;
//  double t1 = 0.0, t2 = 0.0, t3 = 0.0;
//  mwSize H1,W1, H2,W2;
//  mwSize H3,W3, H4,W4;
//#endif // TM
//
//  try {
//    // iterate over each training instance
//    CpyVolConvmatImpl ip = make_initial_CpyVolConvmatImpl( X );
//    mwSize N = X.getSizeAtDim(4);
//
//#ifdef TM
//    mexPrintf("num inst = %d \n", N);
//#endif
//
//    for (mwSize i = 0; i < N; i++) {
//#ifdef TM
//      tmtm.start();
//#endif // TM
//
//      // make phiX: the convolution matrix
//      vol_to_convmat(ip, X, i);
//
//#ifdef TM
//      tmtm.stop();
//      t1 += tmtm.getElapsedTimeInMilliSec();
//#endif // TM
//
//#ifdef TM
//      tmtm.start();
//#endif // TM
//
//      // convolution: Y_ = phiX * F_
//      matw F_ = make_F_();
//      matw Y_ = make_Y_(i);
//      //cu_AxBtoC(convmat, F_, Y_, true); // overwrite Y_ 
//
//#ifdef TM
//      tmtm.stop();
//      t2 += tmtm.getElapsedTimeInMilliSec();
//      H1 = convmat.H; W1 = convmat.W;
//      H2 = F_.H; W2 = F_.W;
//#endif // TM
//
//#ifdef TM
//      tmtm.start();
//#endif // TM
//
//      // plus the bias: Y_ += u * B
//      matw B_ = make_B_();
//      //cu_AxBtoC(u, B_, Y_, false); // accumulation on Y_
//
//#ifdef TM
//      tmtm.stop();
//      t3 += tmtm.getElapsedTimeInMilliSec();
//      H3 =  u.H; W3 = u.W;
//      H4 = B_.H; W4 = B_.W;
//#endif // TM
//    } // for i
//  } // try
//  catch (const blas_ex& e) {
//    free_u();
//    free_convmat();
//    throw conv3d_ex(e.what());
//  }
//
//#ifdef TM
//  tm.stop();
//  double te = tm.getElapsedTimeInMilliSec();
//
//  mexPrintf("conv3d_blas_gpu::vol_to_convmat: %f\n", t1);
//
//  mexPrintf("conv3d_blas_gpu::mut 1: %f\n", t2);
//  mexPrintf("[%d %d] x [%d %d]\n", H1,W1, H2,W2);
//
//  mexPrintf("conv3d_blas_gpu::mut 2: %f\n", t3);
//  mexPrintf("[%d %d] x [%d %d]\n",H3,W3, H4,W4);
//
//  mexPrintf("conv3d_blas_gpu::fprop: %f\n", te);
//#endif // TM
//
//  //free_convmat();
//  //free_u();
//  
//#ifdef TM
//  mexPrintf("\n");
//#endif // TM
//}

//void conv3d_blas_gpu::fprop()
//{
//  create_Y();
//  init_convmat();
//  init_u(); 
//
//#ifdef TM
//  Timer tm;
//  tm.start();
//
//  Timer tmtm;
//  double t1 = 0.0, t2 = 0.0, t3 = 0.0;
//  mwSize H1,W1, H2,W2;
//  mwSize H3,W3, H4,W4;
//
//  hipblasHandle_t hd;
//  hipblasStatus_t st;
//  hipblasCreate(&hd);
//
//#endif // TM
//
//  try {
//    // iterate over each training instance
//    CpyVolConvmatImpl ip = make_initial_CpyVolConvmatImpl( X );
//    mwSize N = X.getSizeAtDim(4);
//
//#ifdef TM
//    mexPrintf("num inst = %d \n", N);
//#endif
//
//    for (mwSize i = 0; i < N; i++) {
//#ifdef TM
//      tmtm.start();
//#endif // TM
//
//      // make phiX: the convolution matrix
//      //vol_to_convmat(ip, X, i);
//
//#ifdef TM
//      tmtm.stop();
//      t1 += tmtm.getElapsedTimeInMilliSec();
//#endif // TM
//
//
//      // convolution: Y_ = phiX * F_
//      matw F_ = make_F_();
//      matw Y_ = make_Y_(i);
//
//      float alpha = 1.0;
//      float beta = 0.0;
//
//#ifdef TM
//      tmtm.start();
//#endif // TM
//
//      st = hipblasSgemm(
//        hd,
//        HIPBLAS_OP_N, HIPBLAS_OP_N,
//        (int)convmat.H, (int)F_.W, (int)convmat.W,
//        &alpha,
//        (float*)convmat.beg, (int)convmat.H,
//        (float*)F_.beg, (int)F_.H,
//        &beta,
//        (float*)Y_.beg, (int)Y_.H);
//
//#ifdef TM
//      tmtm.stop();
//      t2 += tmtm.getElapsedTimeInMilliSec();
//      H1 = convmat.H; W1 = convmat.W;
//      H2 = F_.H; W2 = F_.W;
//#endif // TM
//
//
//      // plus the bias: Y_ += u * B
//      matw B_ = make_B_();
//      
//#ifdef TM
//      tmtm.start();
//#endif // TM
//
//      alpha = 1.0;
//      beta  = 1.0;
//      st = hipblasSgemm(
//        hd,
//        HIPBLAS_OP_N, HIPBLAS_OP_N,
//        (int)u.H, (int)B_.W, (int)u.W,
//        &alpha,
//        (float*)u.beg, (int)u.H,
//        (float*)B_.beg, (int)B_.H,
//        &beta,
//        (float*)Y_.beg, (int)Y_.H);
//
//#ifdef TM
//      tmtm.stop();
//      t3 += tmtm.getElapsedTimeInMilliSec();
//      H3 =  u.H; W3 = u.W;
//      H4 = B_.H; W4 = B_.W;
//#endif // TM
//    } // for i
//  } // try
//  catch (const blas_ex& e) {
//    free_u();
//    free_convmat();
//    throw conv3d_ex(e.what());
//  }
//
//#ifdef TM
//  hipblasDestroy(hd);
//  tm.stop();
//  double te = tm.getElapsedTimeInMilliSec();
//
//  mexPrintf("conv3d_blas_gpu::vol_to_convmat: %f\n", t1);
//
//  mexPrintf("conv3d_blas_gpu::mut 1: %f\n", t2);
//  mexPrintf("[%d %d] x [%d %d]\n", H1,W1, H2,W2);
//
//  mexPrintf("conv3d_blas_gpu::mut 2: %f\n", t3);
//  mexPrintf("[%d %d] x [%d %d]\n",H3,W3, H4,W4);
//
//  mexPrintf("conv3d_blas_gpu::fprop: %f\n", te);
//#endif // TM
//
//  free_convmat();
//  free_u();
//
//#ifdef TM
//  mexPrintf("\n");
//#endif // TM
//
//}

void conv3d_blas_gpu::bprop()
{
  check_X_size();
  create_dX();
  create_dF();
  create_dB();
  init_convmat();
  init_u();

  try {
    // iterate over each instance
    CpyVolConvmatImpl ip = make_initial_CpyVolConvmatImpl( X );
    matw dF_ = make_dF_();
    matw dB_ = make_dB_();
    mwSize N = X.getSizeAtDim(4);
    for (mwSize i = 0; i < N; ++i) {
      // make phiX: the convolution matrix
      vol_to_convmat(ip, X, i);

      // dF += phiX' * dY_
      matw dY_ = make_dY_(i);
      cu_ATxBtoC(convmat, dY_, dF_, false); // accumulation on dF_ TODO: the right cublas

      // dB += u' * dY
      cu_ATxBtoC(u, dY_, dB_, false); // accumulation on dB_

      // dphiX = dY * F'
      matw F_ = make_F_();
      // safe to reuse convmat memory as X and dX have the same size; remember to overwrite it!
      cu_AxBTtoC(dY_, F_, convmat, true);
      // dX(:,:,:,:,i) <-- dphiX
      vol_from_convmat(ip, dX, i);
    }
  }
  catch (const blas_ex& e) {
    free_u();
    free_convmat();
    throw conv3d_ex(e.what());
  }

  free_u();
  free_convmat();
}

//// Impl of helper: fprop
matw conv3d_blas_gpu::make_F_()
{
  matw F_;
  F_.beg = (float*)F.getDataBeg();
  F_.H   = numelVol(F) * F.getSizeAtDim(3);
  F_.W   = F.getSizeAtDim(4);

  return F_;
}

matw conv3d_blas_gpu::make_Y_(mwSize i)
{
  matw Y_;
  Y_.beg = getVolInstDataBeg<float>(Y, i);
  Y_.H   = numelVol(Y);
  Y_.W   = Y.getSizeAtDim(3);

  return Y_;
}

matw conv3d_blas_gpu::make_B_()
{
  matw B_;
  B_.beg = (float*)B.getDataBeg();
  B_.H   = 1;
  B_.W   = numel(B);

  return B_;
}

//// Impl of helper: bprop
matw conv3d_blas_gpu::make_dY_(mwSize i)
{
  matw dY_;
  dY_.beg = getVolInstDataBeg<float>(dY, i);
  dY_.H   = numelVol(dY);
  dY_.W   = dY.getSizeAtDim(3);

  return dY_;
}

matw conv3d_blas_gpu::make_dF_()
{
  matw dF_;
  dF_.beg = (float*)dF.getDataBeg();
  dF_.H   = numelVol(dF) * dF.getSizeAtDim(3);
  dF_.W   = dF.getSizeAtDim(4);

  return dF_;
}

matw conv3d_blas_gpu::make_dB_()
{
  matw dB_;
  dB_.beg = (float*)dB.getDataBeg();
  dB_.H   = 1;
  dB_.W   = numel(dB);
  
  return dB_;
}

//// Impl of helper: the stacked matrix storing phiX or dphiX
conv3d_blas_gpu::CpyVolConvmatImpl conv3d_blas_gpu::make_initial_CpyVolConvmatImpl(const xpuMxArrayTW &vol)
{
  CpyVolConvmatImpl ip;

  ip.vol_i.beg = 0; // to be set later
  for (int i = 0; i < 4; ++i) ip.vol_i.sz[i] = vol.getSizeAtDim(i);

  ip.convmat = this->convmat;

  if ( Y.pa_cpu != 0)
    for (int i = 0; i < 3; ++i) ip.szY[i] = this->Y.getSizeAtDim(i);
  else // dY.pa_cpu != 0
    for (int i = 0; i < 3; ++i) ip.szY[i] = this->dY.getSizeAtDim(i);

  for (int i = 0; i < 3; ++i) ip.szF[i] = this->F.getSizeAtDim(i);
  for (int i = 0; i < 3; ++i) ip.stride[i] = this->stride[i];
  for (int i = 0; i < 6; i++) ip.pad[i] = this->pad[i];

  return ip;
}

void conv3d_blas_gpu::init_convmat()
{
#ifdef TM
  Timer tm;
  tm.start();
#endif // TM

  // set the size
  assert( (Y.pa_cpu != 0) || (dY.pa_cpu != 0) );
  if (Y.pa_cpu != 0) // in FPROP, Y has been set
    convmat.H = numelVol(Y);
  else // (dY != 0), in BPROP, dY has been set
    convmat.H = numelVol(dY);

  convmat.W = numelVol(F) * F.getSizeAtDim(3);
  mwSize nelem = convmat.H * convmat.W;

  // allocate the memory
  void* tmp;
  hipError_t flag = hipMalloc(&tmp,  nelem*sizeof(float) ) ;
  if (flag != hipSuccess) throw conv3d_ex("Out of memory on GPU.\n");
  convmat.beg = (float*)tmp;

  // assures all zeros
  kernelSetZero<float><<<ceil_divide(nelem,CU_NUM_THREADS), CU_NUM_THREADS>>>(convmat.beg, nelem);
  
#ifdef TM
  tm.stop();
  double te = tm.getElapsedTimeInMilliSec();
  mexPrintf("conv3d_blas_gpu::init_convmat: %f\n", te);
#endif // TM

  LOGMSG("conv3d_blas_gpu::init_convmat(): %d KB\n", toKB(nelem, mxSINGLE_CLASS));
}

void conv3d_blas_gpu::free_convmat()
{
//#ifdef TM
//  Timer tm;
//  tm.start();
//#endif // TM

  hipFree( (void*)convmat.beg );

//#ifdef TM
//  tm.stop();
//  double te = tm.getElapsedTimeInMilliSec();
//  mexPrintf("conv3d_blas_gpu::free_convmat: %f\n", te);
//#endif // TM

  LOGMSG("conv3d_blas_gpu::free_convmat()\n");
}

void conv3d_blas_gpu::vol_to_convmat (CpyVolConvmatImpl &ip, xpuMxArrayTW &vol, mwSize iInst)
{
  // set vol(:,:,:,:, i)
  ip.vol_i.beg = getVolInstDataBeg<float>(vol, iInst);

  // do the real job
  mwSize nelem = ip.convmat.H * ip.convmat.W;
  dim3 blkSize( ceil_divide(nelem, CU_NUM_THREADS) );
  dim3 thdSize( CU_NUM_THREADS );
  kernelCpyVolConvmat<DIR_VOL_TO_CONVMAT><<<blkSize, thdSize>>>(ip);
}

void conv3d_blas_gpu::vol_from_convmat(CpyVolConvmatImpl &ip, xpuMxArrayTW &vol, mwSize iInst)
{
  // set vol(:,:,:,:, i)
  ip.vol_i.beg = getVolInstDataBeg<float>(vol, iInst);

  // do the real job
  mwSize nelem = ip.convmat.H * ip.convmat.W;
  dim3 blkSize( ceil_divide(nelem, CU_NUM_THREADS) );
  dim3 thdSize( CU_NUM_THREADS );
  kernelCpyVolConvmat<DIR_VOL_FROM_CONVMAT><<<blkSize, thdSize>>>(ip);
}

void conv3d_blas_gpu::init_u()
{
#ifdef TM
  Timer tm;
  tm.start();
#endif // TM

  // decide the size
  assert( (Y.pa_cpu != 0) || (dY.pa_cpu != 0) );
  if (Y.pa_cpu != 0)
    u.H = numelVol(Y);
  else // (dY != 0)
    u.H = numelVol(dY);

  u.W = 1;
  mwSize nelem = u.H * u.W ;

  // allocate the memory
  void* tmp;
  hipError_t flag = hipMalloc(&tmp, nelem * sizeof(float));
  if (flag != hipSuccess) throw conv3d_ex("Out of memory on GPU.\n");
  u.beg = (float*) tmp;

  // make sure all one
  kernelSetOne<float><<<ceil_divide(nelem,CU_NUM_THREADS), CU_NUM_THREADS>>>(u.beg, nelem);

  LOGMSG("conv3d_blas_gpu::init_u(): %d KB\n", toKB(nelem, mxSINGLE_CLASS));

#ifdef TM
  tm.stop();
  double te = tm.getElapsedTimeInMilliSec();
  mexPrintf("onv3d_blas_gpu::init_u: %f\n", te);
#endif // TM
}

void conv3d_blas_gpu::free_u()
{
#ifdef TM
  Timer tm;
  tm.start();
#endif // TM

  hipFree( (void*)u.beg );

#ifdef TM
  tm.stop();
  double te = tm.getElapsedTimeInMilliSec();
  mexPrintf("conv3d_blas_gpu::free_u: %f\n", te);
#endif // TM

  LOGMSG("conv3d_blas_gpu::free_u()\n");
}